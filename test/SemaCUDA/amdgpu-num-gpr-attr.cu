#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

__attribute__((amdgpu_num_vgpr(64))) // expected-error {{'amdgpu_num_vgpr' attribute only applies to kernel functions}}
__global__ void test_num_vgpr() { }

__attribute__((amdgpu_num_sgpr(32))) // expected-error {{'amdgpu_num_sgpr' attribute only applies to kernel functions}}
__global__ void test_num_sgpr() { }

// expected-error@+2 {{'amdgpu_num_sgpr' attribute only applies to kernel functions}}
// expected-error@+1 {{'amdgpu_num_vgpr' attribute only applies to kernel functions}}
__attribute__((amdgpu_num_sgpr(32), amdgpu_num_vgpr(64)))
__global__ void test_num_vgpr_num_sgpr() { }
