#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

// Check that it's OK for kernels to call HD functions that call device-only
// functions.

#include "Inputs/hip/hip_runtime.h"

__device__ void device_fn(int) {}
// expected-note@-1 {{declared here}}
// expected-note@-2 {{declared here}}

inline __host__ __device__ int hd1() {
  device_fn(0);  // expected-error {{reference to __device__ function}}
  return 0;
}

inline __host__ __device__ int hd2() {
  // No error here because hd2 is only referenced from a kernel.
  device_fn(0);
  return 0;
}

inline __host__ __device__ void hd3(int) {
  device_fn(0);  // expected-error {{reference to __device__ function 'device_fn'}}
}
inline __host__ __device__ void hd3(double) {}

inline __host__ __device__ void hd4(int) {}
inline __host__ __device__ void hd4(double) {
  device_fn(0);  // No error; this function is never called.
}

__global__ void kernel(int) { hd2(); }

template <typename T>
void launch_kernel() {
  kernel<<<0, 0>>>(T());
  hd1();
  hd3(T());
}

void host_fn() {
  launch_kernel<int>();
}
