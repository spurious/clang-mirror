// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -fsyntax-only \
// RUN:   -verify -DEXPECT_ERR %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s

#include <stdarg.h>
#include "Inputs/hip/hip_runtime.h"

__device__ void foo() {
  va_list list;
  va_arg(list, int);
#ifdef EXPECT_ERR
  // expected-error@-2 {{CUDA device code does not support va_arg}}
#endif
}

void bar() {
  va_list list;
  va_arg(list, int);  // OK: host-only
}

__device__ void baz() {
#if !defined(__CUDA_ARCH__)
  va_list list;
  va_arg(list, int);  // OK: only seen when compiling for host
#endif
}

__device__ void vararg(const char* x, ...) {}
// expected-error@-1 {{CUDA device code does not support variadic functions}}

extern "C" __device__ int printf(const char* fmt, ...);  // OK, special case.

// Definition of printf not allowed.
extern "C" __device__ int printf(const char* fmt, ...) { return 0; }
// expected-error@-1 {{CUDA device code does not support variadic functions}}

namespace ns {
__device__ int printf(const char* fmt, ...);
// expected-error@-1 {{CUDA device code does not support variadic functions}}
}
