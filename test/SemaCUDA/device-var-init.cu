#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target

// Make sure we don't allow dynamic initialization for device
// variables, but accept empty constructors allowed by CUDA.

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -I %S/.. -fsyntax-only -verify -o /dev/null %s

// Counterpart in CodeGen covers valid cases that pass Sema
// checks. Here we'll only cover cases that trigger errors.
#include "CodeGenCUDA/device-var-init.cu"

__shared__ int s_v_i = 1;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}

__device__ int d_v_f = f();
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ int s_v_f = f();
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ int c_v_f = f();
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__shared__ T s_t_i = {2};
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}

__device__ EC d_ec_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ EC s_ec_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ EC c_ec_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ EC d_ec_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ EC s_ec_i2 = {3};
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ EC c_ec_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ ETC d_etc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ ETC s_etc_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ ETC c_etc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ ETC d_etc_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ ETC s_etc_i2 = {3};
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ ETC c_etc_i2 = {3};
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ UC d_uc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ UC s_uc;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ UC c_uc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ ECI d_eci;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ ECI s_eci;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ ECI c_eci;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NEC d_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NEC s_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NEC c_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NCV d_ncv;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NCV s_ncv;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NCV c_ncv;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NCF d_ncf;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NCF s_ncf;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NCF c_ncf;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__shared__ NCFS s_ncfs;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}

__device__ UTC d_utc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ UTC s_utc;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ UTC c_utc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ UTC d_utc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ UTC s_utc_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ UTC c_utc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NETC d_netc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NETC s_netc;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NETC c_netc;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ NETC d_netc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ NETC s_netc_i(3);
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ NETC c_netc_i(3);
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ EC_I_EC1 d_ec_i_ec1;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ EC_I_EC1 s_ec_i_ec1;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ EC_I_EC1 c_ec_i_ec1;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ T_V_T d_t_v_t;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_V_T s_t_v_t;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_V_T c_t_v_t;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ T_B_NEC d_t_b_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_B_NEC s_t_b_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_B_NEC c_t_b_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ T_F_NEC d_t_f_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_F_NEC s_t_f_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_F_NEC c_t_f_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ T_FA_NEC d_t_fa_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__shared__ T_FA_NEC s_t_fa_nec;
// expected-error@-1 {{initialization is not supported for __shared__ variables.}}
__constant__ T_FA_NEC c_t_fa_nec;
// expected-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

// Make sure that initialization restrictions do not apply to local
// variables.
__device__ void df_sema() {
  static __shared__ NCFS s_ncfs;
  // expected-error@-1 {{initialization is not supported for __shared__ variables.}}
  static __shared__ UC s_uc;
  // expected-error@-1 {{initialization is not supported for __shared__ variables.}}
  static __device__ int ds;
  // expected-error@-1 {{Within a __device__/__global__ function, only __shared__ variables may be marked "static"}}
  static __constant__ int dc;
  // expected-error@-1 {{Within a __device__/__global__ function, only __shared__ variables may be marked "static"}}
  static int v;
  // expected-error@-1 {{Within a __device__/__global__ function, only __shared__ variables may be marked "static"}}
}
