
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple nvptx64-unknown-unknown -target-cpu sm_60 \
// RUN:            -fcuda-is-device -target-feature +ptx60 \
// RUN:            -S -emit-llvm -o - -x cuda %s \
// RUN:   | FileCheck -check-prefix=CHECK %s
// RUN: %clang_cc1 -triple nvptx-unknown-unknown -target-cpu sm_60 \
// RUN:   -fcuda-is-device -S -o /dev/null -x cuda -verify %s

#define __device__ __attribute__((device))
#define __global__ __attribute__((global))
#define __shared__ __attribute__((shared))
#define __constant__ __attribute__((constant))

// We have to keep all builtins that depend on particular target feature in the
// same function, because the codegen will stop after the very first function
// that encounters an error, so -verify will not be able to find errors in
// subsequent functions.

// CHECK-LABEL: nvvm_sync
__device__ void nvvm_sync(unsigned mask, int i, float f, int a, int b,
                          bool pred) {
  // CHECK: call void @llvm.nvvm.bar.warp.sync(i32
  // expected-error@+1 {{'__nvvm_bar_warp_sync' needs target feature ptx60}}
  __nvvm_bar_warp_sync(mask);
  // CHECK: call void @llvm.nvvm.barrier.sync(i32
  // expected-error@+1 {{'__nvvm_barrier_sync' needs target feature ptx60}}
  __nvvm_barrier_sync(mask);
  // CHECK: call void @llvm.nvvm.barrier.sync.cnt(i32
  // expected-error@+1 {{'__nvvm_barrier_sync_cnt' needs target feature ptx60}}
  __nvvm_barrier_sync_cnt(mask, i);

  //
  // SHFL.SYNC
  //
  // CHECK: call i32 @llvm.nvvm.shfl.sync.down.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_down_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_down_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.down.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_down_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_down_f32(mask, f, a, b);
  // CHECK: call i32 @llvm.nvvm.shfl.sync.up.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_up_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_up_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.up.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_up_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_up_f32(mask, f, a, b);
  // CHECK: call i32 @llvm.nvvm.shfl.sync.bfly.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_bfly_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_bfly_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.bfly.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_bfly_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_bfly_f32(mask, f, a, b);
  // CHECK: call i32 @llvm.nvvm.shfl.sync.idx.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_idx_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_idx_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.idx.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_idx_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_idx_f32(mask, f, a, b);

  //
  // VOTE.SYNC
  //

  // CHECK: call i1 @llvm.nvvm.vote.all.sync(i32
  // expected-error@+1 {{'__nvvm_vote_all_sync' needs target feature ptx60}}
  __nvvm_vote_all_sync(mask, pred);
  // CHECK: call i1 @llvm.nvvm.vote.any.sync(i32
  // expected-error@+1 {{'__nvvm_vote_any_sync' needs target feature ptx60}}
  __nvvm_vote_any_sync(mask, pred);
  // CHECK: call i1 @llvm.nvvm.vote.uni.sync(i32
  // expected-error@+1 {{'__nvvm_vote_uni_sync' needs target feature ptx60}}
  __nvvm_vote_uni_sync(mask, pred);
  // CHECK: call i32 @llvm.nvvm.vote.ballot.sync(i32
  // expected-error@+1 {{'__nvvm_vote_ballot_sync' needs target feature ptx60}}
  __nvvm_vote_ballot_sync(mask, pred);

  // CHECK: ret void
}
