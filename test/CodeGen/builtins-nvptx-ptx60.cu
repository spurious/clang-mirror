
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple nvptx64-unknown-unknown -target-cpu sm_60 \
// RUN:            -fcuda-is-device -target-feature +ptx60 \
// RUN:            -S -emit-llvm -o - -x cuda %s \
// RUN:   | FileCheck -check-prefix=CHECK %s
// RUN: %clang_cc1 -triple nvptx-unknown-unknown -target-cpu sm_60 \
// RUN:   -fcuda-is-device -S -o /dev/null -x cuda -verify %s

#define __device__ __attribute__((device))
#define __global__ __attribute__((global))
#define __shared__ __attribute__((shared))
#define __constant__ __attribute__((constant))

// CHECK-LABEL: nvvm_shfl_sync
__device__ void nvvm_shfl_sync(unsigned mask, int i, float f, int a, int b) {
  // CHECK: call i32 @llvm.nvvm.shfl.sync.down.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_down_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_down_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.down.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_down_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_down_f32(mask, f, a, b);
  // CHECK: call i32 @llvm.nvvm.shfl.sync.up.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_up_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_up_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.up.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_up_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_up_f32(mask, f, a, b);
  // CHECK: call i32 @llvm.nvvm.shfl.sync.bfly.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_bfly_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_bfly_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.bfly.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_bfly_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_bfly_f32(mask, f, a, b);
  // CHECK: call i32 @llvm.nvvm.shfl.sync.idx.i32(i32 {{%[0-9]+}}, i32
  // expected-error@+1 {{'__nvvm_shfl_sync_idx_i32' needs target feature ptx60}}
  __nvvm_shfl_sync_idx_i32(mask, i, a, b);
  // CHECK: call float @llvm.nvvm.shfl.sync.idx.f32(i32 {{%[0-9]+}}, float
  // expected-error@+1 {{'__nvvm_shfl_sync_idx_f32' needs target feature ptx60}}
  __nvvm_shfl_sync_idx_f32(mask, f, a, b);
  // CHECK: ret void
}
