
#include <hip/hip_runtime.h>
// RUN: %clang -v --sysroot=/tmp/no-cuda-there 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --sysroot=%S/Inputs/CUDA 2>&1 | FileCheck %s
// RUN: %clang -v --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// NOCUDA-NOT: Found CUDA installation:
