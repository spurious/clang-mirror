
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// # Check that we properly detect CUDA installation.
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/no-cuda-there 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/Inputs/CUDA 2>&1 | FileCheck %s
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s

// Verify that CUDA include path gets added
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix CUDAINC
// Verify that -nocudainc disables CUDA include paths.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC
// We should not add any CUDA include paths if there's no valid CUDA installation
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// NOCUDA-NOT: Found CUDA installation:

// COMMON: "-triple" "nvptx-nvidia-cuda"
// COMMON-SAME: "-fcuda-is-device"
// CUDAINC-SAME: "-internal-isystem" "{{.*}}/Inputs/CUDA/usr/local/cuda/include"
// NOCUDAINC-NOT: "-internal-isystem" "{{.*}}/cuda/include"
// COMMON-SAME: "-x" "cuda"

