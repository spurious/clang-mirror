
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -std=c++14 -S -verify -fcuda-is-device %s -o /dev/null

// Check how the force_cuda_host_device pragma interacts with template
// instantiations.  The errors here are emitted at codegen, so we can't do
// -fsyntax-only.

template <typename T>
auto foo() {  // expected-note {{declared here}}
  return T();
}

template <typename T>
struct X {
  void foo(); // expected-note {{declared here}}
};

#pragma clang force_cuda_host_device begin
__attribute__((host)) __attribute__((device)) void test() {
  int n = foo<int>();  // expected-error {{reference to __host__ function 'foo<int>'}}
  X<int>().foo();  // expected-error {{reference to __host__ function 'foo'}}
}
#pragma clang force_cuda_host_device end

// Same thing as above, but within a force_cuda_host_device block without a
// corresponding end.

template <typename T>
T bar() {  // expected-note {{declared here}}
  return T();
}

template <typename T>
struct Y {
  void bar(); // expected-note {{declared here}}
};

#pragma clang force_cuda_host_device begin
__attribute__((host)) __attribute__((device)) void test2() {
  int n = bar<int>();  // expected-error {{reference to __host__ function 'bar<int>'}}
  Y<int>().bar();  // expected-error {{reference to __host__ function 'bar'}}
}
