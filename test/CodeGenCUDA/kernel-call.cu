#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -o - | FileCheck %s --check-prefixes=CUDA,CHECK
// RUN: %clang_cc1 -x hip -emit-llvm %s -o - | FileCheck %s --check-prefixes=HIP,CHECK


#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: define{{.*}} void @_Z2g1i
// HIP: call{{.*}}hipSetupArgument
// HIP: call{{.*}}hipLaunchByPtr
// CUDA: call{{.*}}hipSetupArgument
// CUDA: call{{.*}}hipLaunchByPtr
__global__ void g1(int x) {}

// CHECK-LABEL: define{{.*}} i32 @main
int main(void) {
  // HIP: call{{.*}}hipConfigureCall
  // CUDA: call{{.*}}hipConfigureCall
  // CHECK: icmp
  // CHECK: br
  // CHECK: call{{.*}}g1
  g1<<<1, 1>>>(42);
}
